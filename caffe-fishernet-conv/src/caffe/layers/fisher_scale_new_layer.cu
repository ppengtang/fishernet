#include "hip/hip_runtime.h"
#include "caffe/layers/fisher_layer.hpp"

namespace caffe {

template <typename Dtype>
__global__ void FisherScaleNewForwardGPU(const int nthreads,
          const Dtype* bottom_data, const Dtype* bottom_data_weight,
          Dtype* top_data, const int N_, const int inner_num_) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    const int n = (index % (N_ * inner_num_)) / inner_num_;
    top_data[index] = bottom_data[index] * bottom_data_weight[n];
  }
}

template <typename Dtype>
void FisherScaleNewLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  const Dtype* bottom_data_weight = bottom[1]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  const int inner_num_ = bottom[0]->count() / (N_ * outer_num_);
  const int nthreads = bottom[0]->count();
  FisherScaleNewForwardGPU<Dtype><<<CAFFE_GET_BLOCKS(nthreads),
      CAFFE_CUDA_NUM_THREADS>>>(nthreads, bottom_data, bottom_data_weight, top_data, 
                                N_, inner_num_);
}

template <typename Dtype>
__global__ void FisherScaleNewBackwardGPU(const int nthreads,
          const Dtype* top_diff, Dtype* bottom_diff, 
          const Dtype* bottom_data, const int N_, const int inner_num_) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    const int n = (index % (N_ * inner_num_)) / inner_num_;
    bottom_diff[index] = bottom_data[n] * top_diff[index];
  }
}

template <typename Dtype>
__global__ void FisherScaleNewWeightBackwardGPU(const int nthreads,
          const Dtype* top_diff, Dtype* bottom_diff, 
          const Dtype * bottom_data, const int outer_num_, 
          const int N_, const int inner_num_) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    const int n = index;
    bottom_diff[n] = 0;
    for (int i = 0; i < outer_num_; i++) {
      for (int j = 0; j < inner_num_; j++) {
        bottom_diff[n] += bottom_data[i * N_ * inner_num_ + n * inner_num_ + j]
          * top_diff[i * N_ * inner_num_ + n * inner_num_ + j];
      }
    }
  }
}

template <typename Dtype>
void FisherScaleNewLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down,
    const vector<Blob<Dtype>*>& bottom) {
  if (propagate_down[0]) {
    const Dtype* top_diff = top[0]->gpu_diff();
    Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
    const Dtype* bottom_data = bottom[1]->gpu_data();
    const int inner_num_ = bottom[0]->count() / (N_ * outer_num_);
    // Gradient with respect to bottom data
    int nthreads = bottom[0]->count();
    FisherScaleNewBackwardGPU<Dtype><<<CAFFE_GET_BLOCKS(nthreads),
        CAFFE_CUDA_NUM_THREADS>>>(nthreads, top_diff, bottom_diff, bottom_data, N_, inner_num_);
  }
  if (propagate_down[1]) {
    const Dtype* top_diff = top[0]->gpu_diff();
    Dtype* bottom_diff = bottom[1]->mutable_gpu_diff();
    const Dtype* bottom_data = bottom[0]->gpu_data();
    const int inner_num_ = bottom[0]->count() / (N_ * outer_num_);
    // Gradient with respect to bottom data
    int nthreads = N_;
    FisherScaleNewWeightBackwardGPU<Dtype><<<CAFFE_GET_BLOCKS(nthreads),
        CAFFE_CUDA_NUM_THREADS>>>(nthreads, top_diff, bottom_diff, bottom_data, outer_num_, N_, inner_num_);
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(FisherScaleNewLayer);

}  // namespace caffe
