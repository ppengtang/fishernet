#include "hip/hip_runtime.h"
#include "caffe/layers/multi_class_sigmoid_cross_entropy_loss_layer.hpp"

namespace caffe {

template <typename Dtype>
__global__ void MulticlassSigmoidCrossEntropyLossForwardGPU(const int nthreads,
          const Dtype* input_data, const Dtype* label, Dtype* loss) {

  CUDA_KERNEL_LOOP(index, nthreads) {
    loss[index] = log(1 + exp(-input_data[index]))
        + (1.0 - label[index]) * input_data[index];
  }
}

template <typename Dtype>
void MulticlassSigmoidCrossEntropyLossLayer<Dtype>::Forward_gpu(
    const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {
  const Dtype* input_data = bottom[0]->gpu_data();
  const Dtype* label = bottom[1]->gpu_data();
  const int nthreads = batch_size * channels;
  // Since this memory is not used for anything until it is overwritten
  // on the backward pass, we use it here to avoid having to allocate new GPU
  // memory to accumulate intermediate results in the kernel.
  Dtype* loss_data = bottom[0]->mutable_gpu_diff();
  // Similarly, this memory is never used elsewhere, and thus we can use it
  // to avoid having to allocate additional GPU memory.
  // NOLINT_NEXT_LINE(whitespace/operators)
  MulticlassSigmoidCrossEntropyLossForwardGPU<Dtype><<<CAFFE_GET_BLOCKS(nthreads),
      CAFFE_CUDA_NUM_THREADS>>>(nthreads, input_data, label, loss_data);
  Dtype loss;
  caffe_gpu_asum(nthreads, loss_data, &loss);
  loss /= batch_size;
  top[0]->mutable_cpu_data()[0] = loss;
}

template <typename Dtype>
__global__ void MulticlassSigmoidCrossEntropyLossBackwardGPU(const int nthreads, 
		  const Dtype* input_data, const Dtype* label, Dtype* bottom_diff, 
                  const int channels) {

  CUDA_KERNEL_LOOP(index, nthreads) {
    bottom_diff[index] = (1.0 / (1 + exp(-input_data[index])) 
        - label[index]) / channels;
  }
}

template <typename Dtype>
void MulticlassSigmoidCrossEntropyLossLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  if (propagate_down[1]) {
    LOG(FATAL) << this->type()
               << " Layer cannot backpropagate to label inputs.";
  }
  if (propagate_down[0]) {
    Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
    const Dtype* input_data = bottom[0]->gpu_data();
    const Dtype* label = bottom[1]->gpu_data();
    const int nthreads = batch_size * channels;
    
    MulticlassSigmoidCrossEntropyLossBackwardGPU<Dtype><<<CAFFE_GET_BLOCKS(nthreads),
        CAFFE_CUDA_NUM_THREADS>>>(nthreads, input_data, label, bottom_diff,
        channels);
    const Dtype loss_weight = top[0]->cpu_diff()[0];
    
    caffe_gpu_scal(bottom[0]->count(), loss_weight / batch_size, bottom_diff);
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(MulticlassSigmoidCrossEntropyLossLayer);

}  // namespace caffe
